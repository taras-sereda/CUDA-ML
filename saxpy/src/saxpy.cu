#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float* x, float* y) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) y[i] = a * x[i] + y[i];
} 


int main(int argc, char** argv) {
	
	int N = 1<<20; // same as 2^20 or around 1 million. 	
        float *x, *y, *d_x, *d_y;
        x = (float*)malloc(N*sizeof(float));
        y = (float*)malloc(N*sizeof(float));
        
        hipMalloc(&d_x, N*sizeof(float));
        hipMalloc(&d_y, N*sizeof(float));
       
        printf("size of float %lu\n", sizeof(float)); 
        sleep(3);
        printf("sleeping a bit.\n"); 

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<4096, 256>>>(N, 2.0, d_x, d_y);

	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
}
