#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float a, float * restrict x, float * b) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) y[i] = a * x[i] + y[i];
} 


int main(int argc, char** argv) {
	
	int N = 1<<20; // same as 2^20 or around 1 million. 	
	hipMemcpy(d_x, x, N, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N, hipMemcpyHostToDevice);

	saxpy<<4096, 256>>(N, 2.0, d_x, d_y);

	hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);
}
